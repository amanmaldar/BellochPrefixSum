#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

__global__ void gpu_scan_sharemem_phaseI(int *in, int *out, int *aux, int size_tmp, int block_d){

	__shared__ int tmp[2][128];
	unsigned int myblock = blockIdx.x;
	unsigned int tid = threadIdx.x;

	while(myblock < size_tmp){

		tmp[0][tid] = in[myblock*blockDim.x+threadIdx.x];
        __syncthreads();

        int iout = 0;
        for (int d = 0; d < block_d; ++d){   //depth d
        	iout ^= 1;
        	int *tmp_out = tmp[iout];
        	int *tmp_in = tmp[iout ^ 1];
        	int t1 = tid - (1 << d);    //tid - 2^d
        	if (t1 >= 0) {              //tid > 2^d
        		tmp_out[tid] = tmp_in[tid] + tmp_in[t1];
            }
            else {
            	tmp_out[tid] = tmp_in[tid];
            }
            __syncthreads();
        }

        out[myblock*blockDim.x+threadIdx.x] = tmp[iout][tid];
        aux[myblock] = tmp[iout][127]; //last element of each block for next scan
        myblock+=128;
	}
}

__global__ void gpu_scan_sharemem_phaseIII(int *d_out, int *d_out2, int *d_aux2, int size_tmp){

	__shared__ int tmp[128];
	unsigned int myblock = blockIdx.x;
	unsigned int tid = threadIdx.x;

	while(myblock < size_tmp){

		tmp[tid] = d_out[myblock*blockDim.x+threadIdx.x];
        __syncthreads();

        if(myblock > 0){
        	tmp[tid] += d_aux2[myblock-1];
        	__syncthreads();
        }

        d_out2[myblock*blockDim.x+threadIdx.x] = tmp[tid];
        myblock+=128;
	}
}

int main()
{
	int *in;
	int *out;
	int *out_2;
    int *h_out;
	int *aux;
	int *aux_2;
	int num_size = 32000000;
	int size_tmp = (num_size + 127) / 128;
	int block_d = 7;   //depth of block, log2(128) = 7
	hipHostMalloc((void **) &in, sizeof(int)*num_size);
	hipHostMalloc((void **) &out, sizeof(int)*num_size);
    hipHostMalloc((void **) &h_out, sizeof(int)*num_size);
	hipHostMalloc((void **) &out_2, sizeof(int)*num_size);
	hipHostMalloc((void **) &aux, sizeof(int)*size_tmp);  //for checking itermediate values
	hipHostMalloc((void **) &aux_2, sizeof(int)*size_tmp);  //for checking the update itermediate values

	for (int i = 0; i < num_size; i++) {
		in[i] = i;
		out[i] = 0;
    }

    for (int i = 0; i < size_tmp; i++) {
		aux[i] = 0;
    }

    int *d_in;
    int *d_out;
    int *d_aux;
    // int dszp = (num_size)*sizeof(int);
    int dszp_aux = (size_tmp)*sizeof(int);
    hipMalloc((void **) &d_in, sizeof(int)*num_size);
    hipMalloc((void **) &d_out, sizeof(int)*num_size);
    hipMalloc((void **) &d_aux, sizeof(int)*size_tmp);
    // hipMemset(d_out, 0, dszp);
    hipMemset(d_aux, 0, dszp_aux);

// Phase1: scan original data per block and store last element of each block for later scan

    hipMemcpy(d_in, in, sizeof(int)*num_size, hipMemcpyHostToDevice);
    gpu_scan_sharemem_phaseI<<<128, 128>>>(d_in, d_out, d_aux, size_tmp, block_d);
    hipMemcpy(out, d_out, sizeof(int)*num_size, hipMemcpyDeviceToHost);
    hipMemcpy(aux, d_aux, sizeof(int)*size_tmp, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

// Phase2: scan the last element of each block to accumulate the sum of each block

    aux_2[0] = aux[0];
    for(int i = 1; i < size_tmp; i++){
        aux_2[i] = aux[i] + aux_2[i-1];
    }

    int *d_aux2;
    hipMalloc((void **) &d_aux2, sizeof(int)*size_tmp);
    hipMemcpy(d_aux2, aux_2, sizeof(int)*size_tmp, hipMemcpyHostToDevice);

// Phase3: scan the last element of each block to accumulate the sum of each block
    int *d_out2;
    hipMalloc((void **) &d_out2, sizeof(int)*num_size);
    gpu_scan_sharemem_phaseIII<<<128, 128>>>(d_out, d_out2, d_aux2, size_tmp);
    hipMemcpy(out_2, d_out2, sizeof(int)*num_size, hipMemcpyDeviceToHost);

// check the GPU results
    printf("First 200 elements of the scan results:\n");
    for (int i = 0; i < 200; i++){
    	printf("%d\t", out_2[i]);
    	if (i % 10 == 0) printf("\n");
    }
    printf("\n");

    // printf("last 100 elements of the scan results:\n");
    // for (int i = num_size-101; i < num_size-1; i++){
    //     printf("%d\t", out_2[i]);
    //     if (i % 10 == 0) printf("\n");
    // }
    // printf("\n");

// verify via comparasion with cpu version
    int psum = 0;
    for (int i = 0; i < num_size; i++){
        psum += in[i];
        if (psum != out_2[i]) {printf("mismatch at %d, was: %d, should be: %d\n", i, out_2[i], psum); return 1;}
    }
    printf("successfully scan!\n");

    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out2);
    hipFree(d_aux);
    hipFree(d_aux2);


    hipHostFree(in);
    hipHostFree(out);
    hipHostFree(out_2);
    hipHostFree(h_out);
    hipHostFree(aux);
    hipHostFree(aux_2);

    return 0;
}
